#include "hip/hip_runtime.h"


__global__ void addKernel(float *c, const float *a, const float *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

extern "C" void addWithCuda(float *c, const float *a, const float *b, int size)
{
    float *dev_a = 0;
    float *dev_b = 0;
    float *dev_c = 0;

    hipMalloc((void**)&dev_a, size * sizeof(float));
    hipMalloc((void**)&dev_b, size * sizeof(float));
    hipMalloc((void**)&dev_c, size * sizeof(float));

    hipMemcpy(dev_a, a, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size * sizeof(float), hipMemcpyHostToDevice);

    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    hipMemcpy(c, dev_c, size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
}
